#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <fs_constants.h>
#include <fs_calls.cu.h>
#include <host_loop.h>

#define BLOCK_SIZE 32

#define STR_SIZE 256

/* maximum power density possible (say 300W for a 10mm x 10mm chip)	*/
#define MAX_PD	(3.0e6)
/* required precision in degrees	*/
#define PRECISION	0.001
#define SPEC_HEAT_SI 1.75e6
#define K_SI 100
/* capacitance fitting factor	*/
#define FACTOR_CHIP	0.5

#define TLB_SIZE 32

/* chip parameters	*/
float t_chip = 0.0005;
float chip_height = 0.016;
float chip_width = 0.016;
/* ambient temperature, assuming no package at all	*/
float amb_temp = 80.0;

volatile GPUGlobals *gpuGlobals;

void run(int argc, char** argv);

/* define timer macros */
#define pin_stats_reset()   startCycle()
#define pin_stats_pause(cycles)   stopCycle(cycles)
#define pin_stats_dump(cycles)    printf("timer: %Lu\n", cycles)


#define CUDA_CALL_SAFE(f) \
    do \
    {                                                        \
        hipError_t _cuda_error = f;                         \
        if (_cuda_error != hipSuccess)                      \
        {                                                    \
            fprintf(stderr,  \
                "%s, %d, CUDA ERROR: %s %s\n",  \
                __FILE__,   \
                __LINE__,   \
                hipGetErrorName(_cuda_error),  \
                hipGetErrorString(_cuda_error) \
            ); \
            abort(); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)        

static inline double time_diff(struct timeval tv_start, struct timeval tv_end)
{
    return (double)(tv_end.tv_sec - tv_start.tv_sec) * 1000.0 + (double)(tv_end.tv_usec - tv_start.tv_usec) / 1000.0;
}


/*void writeoutput(float *vect, int grid_rows, int grid_cols, char *file)
{
	FILE *fp;

	if ((fp = fopen(file, "wb")) == 0)
    {
        fprintf(stderr, "The file was not opened\n");
        abort();
        exit(EXIT_FAILURE);
    }

    if (fwrite((char *)vect, sizeof(float) * grid_rows * grid_cols, 1, fp) != 1)
    {
        fprintf(stderr, "The file was not written\n");
        abort();
        exit(EXIT_FAILURE);
    }

    fsync(fileno(fp));

	fclose(fp);	
}*/


/*void readinput(float *vect, int grid_rows, int grid_cols, char *file)
{
	FILE *fp;

	if((fp = fopen(file, "rb")) == 0)
    {
        fprintf(stderr, "The file was not opened\n");
        abort();
        exit(EXIT_FAILURE);
    }

    if (fread((char *)vect, sizeof(float) * grid_rows * grid_cols, 1, fp) != 1)
    {
        fprintf(stderr, "The file was not read\n");
        abort();
        exit(EXIT_FAILURE);
    }

	fclose(fp);	
}*/

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))

__global__ void calculate_temp(long iteration,  //number of iteration
                               const char *fn_power,
                               const char *fn_temp_src,
                               const char *fn_temp_dst,
                               /*float *power,   //power input
                               float *temp_src,    //temperature input/output
                               float *temp_dst,    //temperature input/output*/
                               long grid_cols,  //Col of grid
                               long grid_rows,  //Row of grid
							   long border_cols,  // border offset 
							   long border_rows,  // border offset
                               float Cap,      //Capacitance
                               float Rx, 
                               float Ry, 
                               float Rz, 
                               float step, 
                               float time_elapsed,
                               size_t size)
{

    __shared__ TLB<TLB_SIZE> tlb_power;
    __shared__ TLB<TLB_SIZE> tlb_temp_src;
    __shared__ TLB<TLB_SIZE> tlb_temp_dst;

    __shared__ int fd_power;
    __shared__ int fd_temp_src;
    __shared__ int fd_temp_dst;

    __shared__ float temp_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float power_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float temp_t[BLOCK_SIZE][BLOCK_SIZE]; // saving temparary temperature result

	float amb_temp = 80.0;
    float step_div_Cap;
    float Rx_1,Ry_1,Rz_1;
        
	long bx = blockIdx.x;
    long by = blockIdx.y;

	long tx = threadIdx.x;
	long ty = threadIdx.y;
	
	step_div_Cap = step / Cap;
	
	Rx_1 = 1 / Rx;
	Ry_1 = 1 / Ry;
	Rz_1 = 1 / Rz;
	
    // each block finally computes result for a small block
    // after N iterations. 
    // it is the non-overlapping small blocks that cover 
    // all the input data

    // calculate the small block size
	long small_block_rows = BLOCK_SIZE - iteration * 2;//EXPAND_RATE
	long small_block_cols = BLOCK_SIZE - iteration * 2;//EXPAND_RATE

    // calculate the boundary for the block according to 
    // the boundary of its small block
    long blkY = small_block_rows * by - border_rows;
    long blkX = small_block_cols * bx - border_cols;
    long blkYmax = blkY + BLOCK_SIZE - 1;
    long blkXmax = blkX + BLOCK_SIZE - 1;

    // calculate the global thread coordination
	long yidx = blkY + ty;
	long xidx = blkX + tx;

    // load data if it is within the valid input range
	long loadYidx = yidx, loadXidx = xidx;
    long index = grid_cols * loadYidx + loadXidx;

    size = grid_rows * grid_cols * sizeof(float);
       
	if (IN_RANGE(loadYidx, 0, grid_rows - 1) && IN_RANGE(loadXidx, 0, grid_cols - 1))
    {
        fd_power = gopen(fn_power, O_GRDONLY);
        fd_temp_src = gopen(fn_temp_src, O_GRDONLY);

        FatPointer<volatile float, TLB_SIZE> power = gvmmap<volatile float, TLB_SIZE>(NULL, size, 0, O_GRDONLY, fd_power, 0, &tlb_power);
        FatPointer<volatile float, TLB_SIZE> temp_src = gvmmap<volatile float, TLB_SIZE>(NULL, size, 0, O_GRDONLY, fd_temp_src, 0, &tlb_temp_src);

        temp_src += index;
        power += index;

        temp_on_cuda[ty][tx] = *temp_src;  // Load the temperature data from global memory to shared memory
        power_on_cuda[ty][tx] = *power;    // Load the power data from global memory to shared memory

        gclose(fd_power);
        gclose(fd_temp_src);
	}
	__syncthreads();

    // effective range within this block that falls within 
    // the valid range of the input data
    // used to rule out computation outside the boundary.
    long validYmin = (blkY < 0) ? -blkY : 0;
    long validYmax = (blkYmax > grid_rows-1) ? BLOCK_SIZE-1-(blkYmax-grid_rows+1) : BLOCK_SIZE-1;
    long validXmin = (blkX < 0) ? -blkX : 0;
    long validXmax = (blkXmax > grid_cols-1) ? BLOCK_SIZE-1-(blkXmax-grid_cols+1) : BLOCK_SIZE-1;

    long N = ty-1;
    long S = ty+1;
    long W = tx-1;
    long E = tx+1;
    
    N = (N < validYmin) ? validYmin : N;
    S = (S > validYmax) ? validYmax : S;
    W = (W < validXmin) ? validXmin : W;
    E = (E > validXmax) ? validXmax : E;

    bool computed;
    for (long i=0; i<iteration ; i++){ 
        computed = false;
        if( IN_RANGE(tx, i+1, BLOCK_SIZE-i-2) &&  \
              IN_RANGE(ty, i+1, BLOCK_SIZE-i-2) &&  \
              IN_RANGE(tx, validXmin, validXmax) && \
              IN_RANGE(ty, validYmin, validYmax) ) {
              computed = true;
              temp_t[ty][tx] =   temp_on_cuda[ty][tx] + step_div_Cap * (power_on_cuda[ty][tx] + 
                 (temp_on_cuda[S][tx] + temp_on_cuda[N][tx] - 2.0*temp_on_cuda[ty][tx]) * Ry_1 + 
                 (temp_on_cuda[ty][E] + temp_on_cuda[ty][W] - 2.0*temp_on_cuda[ty][tx]) * Rx_1 + 
                 (amb_temp - temp_on_cuda[ty][tx]) * Rz_1);

        }
        __syncthreads();
        if(i==iteration-1)
            break;
        if(computed)	 //Assign the computation range
            temp_on_cuda[ty][tx]= temp_t[ty][tx];
        __syncthreads();
      }

  // update the global memory
  // after the last iteration, only threads coordinated within the 
  // small block perform the calculation and switch on ``computed''
  if (computed){
      fd_temp_dst = gopen(fn_temp_dst, O_GWRONCE);

      FatPointer<volatile float, TLB_SIZE> temp_dst = gvmmap<volatile float, TLB_SIZE>(NULL, size, 0, O_GWRONLY, fd_temp_dst, 0, &tlb_temp_dst);

      temp_dst += index;
      *temp_dst = temp_t[ty][tx];

      gclose(fd_temp_dst);
  }
}

/*
   compute N time steps
*/

int compute_tran_temp(
    /*float *MatrixPower, 
    float *MatrixTemp[2],*/
    const char *fn_power,
    char **fn_temp,
    long col, 
    long row,
    long total_iterations, 
    long num_iterations, 
    long blockCols, 
    long blockRows, 
    long borderCols, 
    long borderRows,
    size_t size) 
{
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 dimGrid(blockCols, blockRows);  
	
	float grid_height = chip_height / row;
	float grid_width = chip_width / col;

	float Cap = FACTOR_CHIP * SPEC_HEAT_SI * t_chip * grid_width * grid_height;
	float Rx = grid_width / (2.0 * K_SI * t_chip * grid_height);
	float Ry = grid_height / (2.0 * K_SI * t_chip * grid_width);
	float Rz = t_chip / (K_SI * grid_height * grid_width);

	float max_slope = MAX_PD / (FACTOR_CHIP * t_chip * SPEC_HEAT_SI);
	float step = PRECISION / max_slope;
	float t;

    float time_elapsed;
	time_elapsed = 0.001;

    int src = 1, dst = 0;
	
	for (t = 0; t < total_iterations; t += num_iterations) 
    {
        int temp = src;
        src = dst;
        dst = temp;
        calculate_temp<<< dimGrid, dimBlock, 0, gpuGlobals->streamMgr->kernelStream >>>(
            MIN(num_iterations, total_iterations-t), 
            /*MatrixPower, 
            MatrixTemp[src], 
            MatrixTemp[dst],*/
            fn_power,
            fn_temp[src],
            fn_temp[dst],
            col, row, borderCols, borderRows, 
            Cap, Rx, Ry, Rz, 
            step, time_elapsed, size);

        run_gpufs_handler(gpuGlobals, 0);
        CUDA_CALL_SAFE(hipStreamSynchronize(gpuGlobals->streamMgr->kernelStream));
	}
    return dst;
}

void usage(int argc, char **argv)
{
	fprintf(stderr, "Usage: %s <grid_rows/grid_cols> <pyramid_height> <sim_time> <temp_file> <power_file> <output_file>\n", argv[0]);
	fprintf(stderr, "\t<grid_rows/grid_cols>  - number of rows/cols in the grid (positive integer)\n");
	fprintf(stderr, "\t<pyramid_height> - pyramid heigh(positive integer)\n");
	fprintf(stderr, "\t<sim_time>   - number of iterations\n");
	fprintf(stderr, "\t<temp_file>  - name of the file containing the initial temperature values of each cell\n");
	fprintf(stderr, "\t<power_file> - name of the file containing the dissipated power values of each cell\n");
	fprintf(stderr, "\t<output_file> - name of the output file\n");
	exit(1);
}

int main(int argc, char** argv)
{
    printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);

    run(argc,argv);

    return EXIT_SUCCESS;
}

void run(int argc, char **argv)
{
    size_t size;
    long grid_rows, grid_cols;
    //float *MatrixTemp[2], *MatrixPower;
    char *tfile, *pfile, *ofile;
    char *fn_power;
    char **fn_temp;
    
    long total_iterations = 60;
    long pyramid_height = 1; // number of iterations

    struct timeval tv_start, tv_end;
    struct timeval tv_start_total, tv_end_total;
    double total_time = 0;       // in ms
    double kernel_time = 0;       // in ms
	
	if (argc != 7)
		usage(argc, argv);
	if((grid_rows = atol(argv[1]))<=0||
	   (grid_cols = atol(argv[1]))<=0||
       (pyramid_height = atoi(argv[2]))<=0||
       (total_iterations = atoi(argv[3]))<=0)
		usage(argc, argv);
		
    gettimeofday(&tv_start_total, NULL);

	tfile = argv[4];
    pfile = argv[5];
    ofile = argv[6];

    initializer(&gpuGlobals);

    fn_temp = (char **)malloc(sizeof(char *) * 2);

    size_t str_size = sizeof(char) * (strlen(pfile) + 1);
    CUDA_CALL_SAFE(hipMalloc(&fn_power, str_size));
    CUDA_CALL_SAFE(hipMemcpy(fn_power, pfile, str_size, hipMemcpyHostToDevice));

    str_size = sizeof(char) * (strlen(tfile) + 1);
    CUDA_CALL_SAFE(hipMalloc(&fn_temp[0], str_size));
    CUDA_CALL_SAFE(hipMemcpy(fn_temp[0], tfile, str_size, hipMemcpyHostToDevice));

    str_size = sizeof(char) * (strlen(ofile) + 1);
    CUDA_CALL_SAFE(hipMalloc(&fn_temp[1], str_size));
    CUDA_CALL_SAFE(hipMemcpy(fn_temp[1], ofile, str_size, hipMemcpyHostToDevice));

	
    size = grid_rows * grid_cols * sizeof(float);

    /* --------------- pyramid parameters --------------- */
    # define EXPAND_RATE 2// add one iteration will extend the pyramid base by 2 per each borderline
    long borderCols = (pyramid_height)*EXPAND_RATE/2;
    long borderRows = (pyramid_height)*EXPAND_RATE/2;
    long smallBlockCol = BLOCK_SIZE-(pyramid_height)*EXPAND_RATE;
    long smallBlockRow = BLOCK_SIZE-(pyramid_height)*EXPAND_RATE;
    long blockCols = grid_cols/smallBlockCol+((grid_cols%smallBlockCol==0)?0:1);
    long blockRows = grid_rows/smallBlockRow+((grid_rows%smallBlockRow==0)?0:1);

    /*CUDA_CALL_SAFE(hipMallocManaged((void **)&MatrixTemp[0], sizeof(float) * size));
    CUDA_CALL_SAFE(hipMallocManaged((void **)&MatrixTemp[1], sizeof(float) * size));
    CUDA_CALL_SAFE(hipMallocManaged((void **)&MatrixPower, sizeof(float) * size));*/

    printf("pyramidHeight: %d\ngridSize: [%d, %d]\nborder:[%d, %d]\nblockGrid:[%d, %d]\ntargetBlock:[%d, %d]\n",\
        pyramid_height, grid_cols, grid_rows, borderCols, borderRows, blockCols, blockRows, smallBlockCol, smallBlockRow);
	
    printf("Start computing the transient temperature\n");

    gettimeofday(&tv_start, NULL);
    int ret = compute_tran_temp(
        /*MatrixPower,
        MatrixTemp,*/
        fn_power,
        fn_temp,
        grid_cols, grid_rows,
	    total_iterations,pyramid_height, blockCols, blockRows, borderCols, borderRows, size);
    gettimeofday(&tv_end, NULL);
    kernel_time += time_diff(tv_start, tv_end);

    CUDA_CALL_SAFE(hipFree(fn_temp[0]));
    CUDA_CALL_SAFE(hipFree(fn_temp[1]));
    CUDA_CALL_SAFE(hipFree(fn_power));
    free(fn_temp);

    gettimeofday(&tv_end_total, NULL);
    total_time += time_diff(tv_start_total, tv_end_total);

    printf("==> header: kernel_time (ms),total_time (ms)\n");
    printf("==> data: %f,%f\n", kernel_time, total_time);
}
