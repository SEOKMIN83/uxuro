#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <string.h>
#include <unistd.h>

#include <hip/hip_runtime_api.h>

//#include "./util/num/num.h"

#include "lavaMD.h"
#include "cudaio.h"
#include "timer.h"

#include "kernel_gpu_cuda.cu"

static void
kernel_gpu_cuda_wrapper(par_str par_cpu, dim_str dim_cpu,
			cuio_ptr_t ptr_box, cuio_ptr_t ptr_rv,
			cuio_ptr_t ptr_qv, cuio_ptr_t ptr_fv)
{
	dim3	threads;
	dim3	blocks;

	blocks.x = dim_cpu.number_boxes;
	blocks.y = 1;
	threads.x = NUMBER_THREADS;			// define the number of threads in the block
	threads.y = 1;

	// launch kernel - all boxes
	kernel_gpu_cuda<<<blocks, threads>>>(par_cpu, dim_cpu, (box_str *)ptr_box.ptr_d, (FOUR_VECTOR *)ptr_rv.ptr_d, (fp_t *)ptr_qv.ptr_d, (FOUR_VECTOR *)ptr_fv.ptr_d);
	getLastCudaError("kernel_gpu_cuda() execution failed\n");
	checkCudaErrors(hipDeviceSynchronize());
}

static void
confer_load(FILE *fp, const char *fpath, void *ctx)
{
	char	buf[1024];
	int	*pn_boxes = (int *)ctx;

	if (fgets(buf, 1024, fp) == NULL) {
		fprintf(stderr, "cannot get # of boxes: %s\n", fpath);
		exit(2);
	}
	if (sscanf(buf, "%d", pn_boxes) != 1) {
		fprintf(stderr, "invalid format: %s\n", fpath);
		exit(3);
	}
}

int 
main(int argc, char *argv [])
{
	// system memory
	par_str	par_cpu;
	dim_str	dim_cpu;
	cuio_ptr_t	ptr_box;
	cuio_ptr_t	ptr_rv;
	cuio_ptr_t	ptr_qv;
	cuio_ptr_t	ptr_fv;
	unsigned	ticks_pre, ticks_kern, ticks_post;
	char	*folder;

	printf("thread block size of kernel = %d \n", NUMBER_THREADS);

	if (argc != 2) {
		fprintf(stderr, "Usage: %s <folder>\n", argv[0]);
		abort();
		exit(EXIT_FAILURE);
	}

	folder = argv[1];

	dim_cpu.cur_arg = 1;

	cuio_init(CUIO_TYPE_NONE, folder);
	cuio_load_conf(confer_load, &dim_cpu.boxes1d_arg);

	if (dim_cpu.boxes1d_arg < 0) {
		fprintf(stderr, "ERROR: wrong # of boxes configuration\n");
		abort();
		exit(EXIT_FAILURE);
	}

	// Print configuration
	printf("Configuration used: boxes1d = %d\n", dim_cpu.boxes1d_arg);

	par_cpu.alpha = 0.5;

	// total number of boxes
	dim_cpu.number_boxes = dim_cpu.boxes1d_arg * dim_cpu.boxes1d_arg * dim_cpu.boxes1d_arg;

	// how many particles space has in each direction
	dim_cpu.space_elem = dim_cpu.number_boxes * NUMBER_PAR_PER_BOX;
	dim_cpu.space_mem = dim_cpu.space_elem * sizeof(FOUR_VECTOR);
	dim_cpu.space_mem2 = dim_cpu.space_elem * sizeof(fp_t);

	// box array
	dim_cpu.box_mem = dim_cpu.number_boxes * sizeof(box_str);

	init_tickcount();

	ptr_box = cuio_load_floats("box.mem", dim_cpu.box_mem / sizeof(float), CUIO_MODE_READONLY);
	ptr_rv = cuio_load_floats("rv.mem", dim_cpu.space_mem / sizeof(float), CUIO_MODE_READONLY);
	ptr_qv = cuio_load_floats("qv.mem", dim_cpu.space_elem, CUIO_MODE_READONLY);
	ptr_fv = cuio_load_floats("fv.mem", dim_cpu.space_mem / sizeof(float), CUIO_MODE_WRITEONLY);

	cuio_memcpy_h2d(&ptr_box);
	cuio_memcpy_h2d(&ptr_rv);
	cuio_memcpy_h2d(&ptr_qv);

	ticks_pre = get_tickcount();
	
	init_tickcount();
	kernel_gpu_cuda_wrapper(par_cpu, dim_cpu, ptr_box, ptr_rv, ptr_qv, ptr_fv);
	ticks_kern = get_tickcount();

	init_tickcount();

	cuio_memcpy_d2h(&ptr_fv);
	cuio_unload_floats("fv.mem", &ptr_fv);
	cuio_free_mem(&ptr_box);
	cuio_free_mem(&ptr_rv);
	cuio_free_mem(&ptr_qv);

	ticks_post = get_tickcount();

	printf("pre time(us): %u\n", ticks_pre);
	printf("kernel time(us): %u\n", ticks_kern);
	printf("post time(us): %u\n", ticks_post);

	return 0;
}
