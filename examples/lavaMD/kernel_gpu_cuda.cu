#include "hip/hip_runtime.h"
static __global__ void
kernel_gpu_cuda(par_str d_par_gpu, dim_str d_dim_gpu, box_str *d_box_gpu,
		FOUR_VECTOR *d_rv_gpu, fp_t *d_qv_gpu, FOUR_VECTOR *d_fv_gpu)
{
	int	bx = blockIdx.x;	// get current horizontal block index (0-n)
	int	tx = threadIdx.x;	// get current horizontal thread index (0-n)
	int	wtx = tx;

	if (bx < d_dim_gpu.number_boxes) {
		// parameters
		fp_t	a2 = 2.0*d_par_gpu.alpha*d_par_gpu.alpha;

		// home box
		int	first_i;
		FOUR_VECTOR	*rA;
		FOUR_VECTOR	*fA;
		__shared__ FOUR_VECTOR	rA_shared[100];

		// nei box
		int	pointer;
		int	k = 0;
		int	first_j;
		FOUR_VECTOR	*rB;
		fp_t	*qB;
		int	j = 0;
		__shared__ FOUR_VECTOR	rB_shared[100];
		__shared__ double	qB_shared[100];

		// common
		fp_t r2, u2, vij, fs, fxij, fyij, fzij;
		THREE_VECTOR	d;

		// home box - box parameters
		first_i = d_box_gpu[bx].offset;

		// home box - distance, force, charge and type parameters
		rA = &d_rv_gpu[first_i];
		fA = &d_fv_gpu[first_i];

		// home box - shared memory
		while (wtx < NUMBER_PAR_PER_BOX) {
			rA_shared[wtx] = rA[wtx];
			wtx = wtx + NUMBER_THREADS;
		}
		wtx = tx;

		// synchronize threads  - not needed, but just to be safe
		__syncthreads();

		//------------------------------
		//	nei box loop
		//------------------------------

		// loop over neiing boxes of home box
		for (k = 0; k < (1 + d_box_gpu[bx].nn); k++) {
			//	nei box - get pointer to the right box
			if (k == 0) {
				pointer = bx;	// set first box to be processed to home box
			}
			else {
				pointer = d_box_gpu[bx].nei[k - 1].number;	// remaining boxes are nei boxes
			}

			//	Setup parameters

			// nei box - box parameters
			first_j = d_box_gpu[pointer].offset;

			// nei box - distance, (force), charge and (type) parameters
			rB = &d_rv_gpu[first_j];
			qB = &d_qv_gpu[first_j];

			//	Setup parameters

			// nei box - shared memory
			while (wtx < NUMBER_PAR_PER_BOX) {
				rB_shared[wtx] = rB[wtx];
				qB_shared[wtx] = qB[wtx];
				wtx = wtx + NUMBER_THREADS;
			}
			wtx = tx;

			// synchronize threads because in next section each thread accesses data brought in by different threads here
			__syncthreads();

			//	Calculation

			// loop for the number of particles in the home box
			while (wtx < NUMBER_PAR_PER_BOX) {
				// loop for the number of particles in the current nei box
				for (j = 0; j < NUMBER_PAR_PER_BOX; j++) {
					r2 = (fp_t)rA_shared[wtx].v + (fp_t)rB_shared[j].v - DOT((fp_t)rA_shared[wtx], (fp_t)rB_shared[j]); 
					u2 = a2 * r2;
					vij = exp(-u2);
					fs = 2*vij;

					d.x = (fp_t)rA_shared[wtx].x  - (fp_t)rB_shared[j].x;
					fxij=fs*d.x;
					d.y = (fp_t)rA_shared[wtx].y  - (fp_t)rB_shared[j].y;
					fyij=fs*d.y;
					d.z = (fp_t)rA_shared[wtx].z  - (fp_t)rB_shared[j].z;
					fzij = fs * d.z;

					fA[wtx].v +=  (double)((fp_t)qB_shared[j] * vij);
					fA[wtx].x +=  (double)((fp_t)qB_shared[j] * fxij);
					fA[wtx].y +=  (double)((fp_t)qB_shared[j] * fyij);
					fA[wtx].z +=  (double)((fp_t)qB_shared[j] * fzij);
				}

				// increment work thread index
				wtx = wtx + NUMBER_THREADS;
			}

			// reset work index
			wtx = tx;

			// synchronize after finishing force contributions from current nei box not to cause conflicts when starting next box
			__syncthreads();
		}
	}
}
